#include "hip/hip_runtime.h"
// Adapted from https://github.com/HandH1998/QQQ

/*
 * Adapted from https://github.com/IST-DASLab/marlin/blob/master/marlin/marlin_cuda_kernel.cu
 * https://github.com/IST-DASLab/marlin/blob/master/marlin/marlin_cuda.cpp
 * Modified by HandH1998
 * Copyright (C) 2024 HandH1998
 * Copyright (C) Marlin.2024 Elias Frantar (elias.frantar@ist.ac.at)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>


constexpr int ceildiv(int a, int b) {
  return (a + b - 1) / b;
}

// Instances of `Vec` are used to organize groups of >>registers<<, as needed for instance as inputs to tensor core
// operations. Consequently, all corresponding index accesses must be compile-time constants, which is why we
// extensively use `#pragma unroll` throughout the kernel code to guarantee this.
template <typename T, int n>
struct Vec {
  T elems[n];
  __device__ T& operator[](int i) {
    return elems[i];
  }
};

using I4 = Vec<int, 4>;

// Matrix fragments for tensor core instructions; their precise layout is documented here: 
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#matrix-fragments-for-mma-m16n8k16-with-integer-type
using FragA = Vec<uint32_t, 2>;
using FragB = Vec<uint32_t, 1>;
using FragC = Vec<int, 4>;
using FragS_GROUP = Vec<half2, 1>; // weight per-group quantization scales
using FragS_CHANNEL = Vec<float, 2>; // weight per-channel quantization scales or activation per-token quantization scales

// Predicated asynchronous global->shared copy; used for inputs A where we apply predication to handle batchsizes that
// are not multiples of 16.
__device__ inline void cp_async4_pred(void* smem_ptr, const void* glob_ptr, bool pred = true) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "{\n"
    "   .reg .pred p;\n"
    "   setp.ne.b32 p, %0, 0;\n"
    "   @p cp.async.cg.shared.global [%1], [%2], %3;\n"
    "}\n" :: "r"((int) pred), "r"(smem), "l"(glob_ptr), "n"(BYTES)
  );
}

// Asynchronous global->shared copy
__device__ inline void cp_async4(void* smem_ptr, const void* glob_ptr) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
      "{\n"
      "   cp.async.cg.shared.global [%0], [%1], %2;\n"
      "}\n" ::"r"(smem),
      "l"(glob_ptr), "n"(BYTES));
}

// NOTE(HandH1998): cp.async.cg only support BYTES = 16, however,
// cp.async.ca can support BYTES = 4, 8, 16;
// as s1's shape is equal to prob_m, we need set s1 to float type,
// and cp_size = 1 float, i.e., 4 BYTES
// Asynchronous global->shared copy for activation quantizaton scales s1
__device__ inline void cp_async1(void* smem_ptr, const void* glob_ptr) {
  const int BYTES = 4;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
      "{\n"
      "   cp.async.ca.shared.global [%0], [%1], %2;\n"
      "}\n" ::"r"(smem),
      "l"(glob_ptr), "n"(BYTES));
}

// Async copy fence.
__device__ inline void cp_async_fence() {
  asm volatile("cp.async.commit_group;\n" ::);
}

// Wait until at most `n` async copy stages are still pending.
template <int n>
__device__ inline void cp_async_wait() {
  asm volatile("cp.async.wait_group %0;\n" :: "n"(n));
}

// m16n8k16 tensor core mma instruction with int8 inputs and int32 output/accumulation.
__device__ inline void mma(const FragA& a_frag, const FragB& frag_b, FragC& frag_c) {
  const uint32_t* a = reinterpret_cast<const uint32_t*>(&a_frag);
  const uint32_t* b = reinterpret_cast<const uint32_t*>(&frag_b);
  int* c = reinterpret_cast<int*>(&frag_c);
  asm volatile(
    "mma.sync.aligned.m16n8k16.row.col.satfinite.s32.s8.s8.s32 "
    "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
    : "=r"(c[0]), "=r"(c[1]), "=r"(c[2]), "=r"(c[3])
    :  "r"(a[0]),  "r"(a[1]),  "r"(b[0]),
       "r"(c[0]),  "r"(c[1]),  "r"(c[2]),  "r"(c[3])
  );
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared memory, directly in tensor core layout.
__device__ inline void ldsm4(FragA& frag_a, const void* smem_ptr) {
  uint32_t* a = reinterpret_cast<uint32_t*>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0,%1}, [%2];\n"
    : "=r"(a[0]), "=r"(a[1]) : "r"(smem)
  );
}

inline __device__ half2 float2_to_half2(float2 f) {
  uint32_t res;
  // NOTE(HandH1998): h0,h1 should be uint16_t, not half
  uint16_t h0, h1;
  asm volatile("cvt.rn.f16.f32 %0, %1;\n" : "=h"(h0) : "f"(f.x));
  asm volatile("cvt.rn.f16.f32 %0, %1;\n" : "=h"(h1) : "f"(f.y));
  asm volatile("mov.b32 %0, {%1, %2};\n" : "=r"(res) : "h"(h0), "h"(h1));
  return reinterpret_cast<half2&>(res);
}

inline __device__ float int32_to_float(int h) {
  float res;
  asm volatile("cvt.rn.f32.s32 %0, %1;\n" : "=f"(res) : "r"(h));
  return res;
}

// Efficiently dequantize an int32 value into a full B-fragment of 4 int8 values for weight per channel dequant.
__device__ inline FragB dequant_per_channel(int q) {
  static constexpr int MASK = 0xf0f0f0f0;
  FragB frag_b;
  frag_b[0] = (q & MASK);
  return frag_b;
}

// Lookup-table based 3-input logical operation; explicitly used for dequantization as the compiler does not seem to
// automatically recognize it in all cases. 
template <int lut>
__device__ inline uint32_t lop3(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t res;
  asm volatile(
    "lop3.b32 %0, %1, %2, %3, %4;\n"
    : "=r"(res) : "r"(a), "r"(b), "r"(c), "n"(lut)
  );
  return res;
}

// TODO(HandH1998): optimize dequant_per_group, as it doesn't have a very good performance for now
// Efficiently dequantize an int32 value into a full B-fragment of 4 int8 values for weight per group dequant.
__device__ inline FragB dequant_per_group(int q, FragS_GROUP& frag_s, int i) {
  // convert 4 int8 to 4 half
  static constexpr uint32_t LO = 0x000f000f;
  static constexpr uint32_t HI = 0x00f000f0;
  static constexpr uint32_t EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  uint32_t t0 = lop3<(0xf0 & 0xcc) | 0xaa>(q, LO, EX);
  uint32_t t1 = lop3<(0xf0 & 0xcc) | 0xaa>(q, HI, EX);
  // We want signed int4 outputs, hence we fuse the `-8` symmetric zero point directly into `SUB` and `ADD`.
  static constexpr uint32_t SUB = 0x64086408;
  static constexpr uint32_t MUL = 0x2c002c00;
  static constexpr uint32_t ADD = 0xd480d480;
  *reinterpret_cast<half2*>(&t0) = __hsub2(
    *reinterpret_cast<half2*>(&t0),
    *reinterpret_cast<const half2*>(&SUB)
  );
  *reinterpret_cast<half2*>(&t1) = __hfma2(
    *reinterpret_cast<half2*>(&t1),
    *reinterpret_cast<const half2*>(&MUL), *reinterpret_cast<const half2*>(&ADD)
  );

  uint16_t s = reinterpret_cast<uint16_t*>(&frag_s)[i];
  uint32_t double_s;
  // pack 2xfp16 to half2
  asm volatile("mov.b32 %0, {%1, %2};\n" : "=r"(double_s) : "h"(s), "h"(s));
  // dequant and convert 4 half to 4 uint8 (be placed at the low 8 bits of 4 half, respectively)
  static constexpr uint32_t MAGIC_NUM = 0x64806480;
  *reinterpret_cast<half2*>(&t0) = __hfma2(
    *reinterpret_cast<half2*>(&t0),
    *reinterpret_cast<half2*>(&double_s), *reinterpret_cast<const half2*>(&MAGIC_NUM)
  );
  *reinterpret_cast<half2*>(&t1) = __hfma2(
    *reinterpret_cast<half2*>(&t1),
    *reinterpret_cast<half2*>(&double_s), *reinterpret_cast<const half2*>(&MAGIC_NUM)
  );
  // take out the 4 uint8 from 4 half, then convert them to 4 int8 and pack 4 int8 into 1 uint32
  FragB frag_b;
  uint32_t uint8s;
  static constexpr uint32_t MASK_0246     = 0x6420;
  static constexpr uint32_t UINT8s_TO_INT8s_MASK    = 0x80808080;
  asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(uint8s) : "r"(t0), "r"(t1), "n"(MASK_0246));
  frag_b[0] = (uint8s ^ UINT8s_TO_INT8s_MASK);
  return frag_b;
}

// Wait until barrier reaches `count`, then lock for current threadblock.
__device__ inline void barrier_acquire(int* lock, int count) {
  if (threadIdx.x == 0) {
    int state = -1;
    do
      // Guarantee that subsequent writes by this threadblock will be visible globally.
      asm volatile ("ld.global.acquire.gpu.b32 %0, [%1];\n" : "=r"(state) : "l"(lock));
    while (state != count);
  }
  __syncthreads();
}

// Release barrier and increment visitation count.
__device__ inline void barrier_release(int* lock, bool reset = false) {
  __syncthreads();
  if (threadIdx.x == 0) {
    if (reset) {
      lock[0] = 0;
      return;
    }
    int val = 1;
    // Make sure that all writes since acquiring this barrier are visible globally, while releasing the barrier. 
    asm volatile ("fence.acq_rel.gpu;\n");
    asm volatile ("red.relaxed.gpu.global.add.s32 [%0], %1;\n" : : "l"(lock), "r"(val)); 
  }
}


template <
  const int threads, // number of threads in a threadblock
  const int thread_m_blocks, // number of 16x16 blocks in the m dimension (batchsize) of the threadblock 
  const int thread_n_blocks, // same for n dimension (output) 
  const int thread_k_blocks, // same for k dimension (reduction)
  const int stages, // number of stages for the async global->shared fetch pipeline
  const int group_blocks = -1 // number of consecutive 16x16 blocks with a separate quantization scale
>
__global__ void Marlin(
  const int4* __restrict__ A, // int8 input matrix of shape mxk 
  const int4* __restrict__ B, // 4bit quantized weight matrix of shape kxn 
        int4* __restrict__ C, // int32 global_reduce buffer of shape (max_par*16*4)xn , as int8 tensor core's output is int32 dtype
        int4* __restrict__ D, // fp16 output buffer of shape mxn
  const float* __restrict__ s1, // fp32 activation per-token quantization scales of shape mx1
  const int4* __restrict__ s2, // fp32 weight per-channel quantization scales of shape 1xn 
  const int4* __restrict__ s3, // fp16 weight per-group quantization scales of shape (k/groupsize)xn, when group_blocks=-1, it should be nullptr
  int  prob_m, // batch dimension m
  int  prob_n, // output dimension n
  int  prob_k, // reduction dimension k
  int* locks // extra global storage for barrier synchronization 
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the same size, which might involve multiple 
  // column "slices" (of width 16 * `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM example: 
  //   0 1 3 
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it ensures good utilization of all SMs
  // for many kinds of shape and GPU configurations, while requiring as few slow global cross-threadblock reductions as 
  // possible.
  
  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);

  // Ensure that the number of tiles in each stripe is a multiple of the groupsize; this avoids an annoying special case
  // where a stripe starts in the middle of group.
  if constexpr (group_blocks != -1)
    iters = (group_blocks / thread_k_blocks) * ceildiv(iters, (group_blocks / thread_k_blocks));

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters; // number of threadblock tiles in the current slice
  int slice_count = 0; // total number of active threadblocks in the current slice
  int slice_idx; // index of threadblock in current slice; numbered bottom to top

  // We can easily implement parallel problem execution by just remapping indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 16;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 4;
    D += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    s1 += (slice_col_par / n_tiles) * 16 * thread_m_blocks;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for synchronization.
  auto init_slice = [&] () {
    slice_iters = iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel)
      slice_iters = 0;
    if (slice_iters == 0)
      return;
    if (slice_row + slice_iters > k_tiles) 
      slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0)
        slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0)
          slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 16;
      C += 16 * thread_m_blocks * prob_n / 4;
      D += 16 * thread_m_blocks * prob_n / 8;
      s1 += 16 * thread_m_blocks;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  int a_gl_stride = prob_k / 16; // stride of the A matrix in global memory
  // We typically use `constexpr` to indicate that this value is a compile-time constant
  constexpr int a_sh_stride = 16 * thread_k_blocks / 16; // stride of an A matrix tile in shared memory
  constexpr int a_gl_rd_delta_o = 16 * thread_k_blocks / 16; // delta between subsequent A tiles in global memory
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o); // between subsequent accesses within a tile
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o); // between shared memory writes
  constexpr int a_sh_rd_delta_o = 1 * ((threads / 32) / (thread_n_blocks / 4)); // between shared memory tile reads
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16; // within a shared memory tile
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks); // overall size of a tile
  constexpr int a_sh_wr_iters = ceildiv(a_sh_stage, a_sh_wr_delta); // number of shared write iterations for a tile

  int b_gl_stride = 16 * prob_n / 32;
  constexpr int b_sh_stride = 32 * thread_n_blocks / 4;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride);
  constexpr int b_sh_wr_delta = threads;
  constexpr int b_sh_rd_delta = threads;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  constexpr int s1_sh_stride = 16 * thread_m_blocks;

  constexpr int s2_sh_stride = 16 * thread_n_blocks / 4;

  int s3_gl_stride = prob_n / 8;
  constexpr int s3_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s3_sh_stage = s3_sh_stride;
  int s3_gl_rd_delta = s3_gl_stride;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  // NOTE(HandH1998): int8 input a only need 16 threads to load 16x16 matrix
  int a_sh_rd = a_sh_stride * ((threadIdx.x % 32) % 16);
  a_sh_rd += 1 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x;
  int b_sh_rd = threadIdx.x;

  int s1_gl_rd = threadIdx.x;
  // NOTE(HandH1998): activation scale s1 need shuffle to [0, 8, 1, 9, 2, 10, 3, 11, 4, 12, 5, 13, 6, 14, 7, 15]
  // for example, 0, 8 row scales serve for thread 0, 1, 2, 3. For more details, refer to mma operand A layout
  // as s1's size is not fixed, we can not shuffle before inference
  // we shuffle it when fetching s1 from global memory to shared memory, that's why s1_sh_wr is like this
  int s1_sh_wr = (threadIdx.x / 16) * 16 + (threadIdx.x % 8) * 2 + (threadIdx.x % 16) / 8;
  int s1_sh_rd = (threadIdx.x % 32) / 4;
  bool s1_sh_wr_pred = threadIdx.x < prob_m;

  int s2_gl_rd = s2_sh_stride * slice_col + threadIdx.x;
  int s2_sh_wr = threadIdx.x;
  int s2_sh_rd = 16 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + 2 * ((threadIdx.x % 32) % 4);
  bool s2_sh_wr_pred = threadIdx.x < s2_sh_stride;

  int s3_gl_rd, s3_sh_wr, s3_sh_rd;
  bool s3_sh_wr_pred;
  if constexpr (group_blocks != -1) {
    s3_gl_rd = s3_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) + s3_sh_stride * slice_col + threadIdx.x;
    s3_sh_wr = threadIdx.x;
    // NOTE(HandH1998): s3_sh_rd is related to mma output C
    s3_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + (threadIdx.x % 32) / 4;
    s3_sh_wr_pred = threadIdx.x < s3_sh_stride;
  }

  // Precompute which thread should not read memory in which iterations; this is needed if there are more threads than
  // required for a certain tilesize or when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;

  // To ensure that writing and reading A tiles to/from shared memory, the latter in fragment format, is fully bank
  // conflict free, we need to use a rather fancy XOR-based layout. The key here is that neither reads nor writes of 
  // the 16-byte `int4` blocks of 8 consecutive threads involve the same shared memory banks. Further, it seems (based
  // on NSight-Compute) that each warp must also write a consecutive memory segment?
  auto transform_a = [&] (int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main loop unrolls, all shared memory 
  // accesses are static, we simply precompute both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
    #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] = transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd); 
  }

  // Since B-accesses have non-constant stride they have to be computed at runtime; we break dependicies between
  // subsequent accesses with a tile by maintining multiple pointers (we have enough registers), a tiny optimization.
  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines. 
  // NOTE(HandH1998): stages need >= 4, otherwise, sh_s1 = sh + max(stages * a_sh_stage + stages * b_sh_stage, 4 * stages * a_sh_stage)
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_s1 = sh_b + (stages * b_sh_stage);
  int4* sh_s2 = sh_s1 + s1_sh_stride;
  int4* sh_s3 = sh_s2 + s2_sh_stride;

  // Register storage for double buffer of shared memory reads. 
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2];
  FragC frag_c[thread_m_blocks][4][2];
  FragS_GROUP frag_s3[2][4];
  FragS_CHANNEL frag_s1[thread_m_blocks];
  FragS_CHANNEL frag_s2[2][4];

  // Zero accumulators.
  auto zero_accums = [&] () {
    #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<int*>(frag_c)[i] = 0;
  };

  // Asynchronously fetch the next A, B and s tile from global to the next shared memory pipeline location.
  auto fetch_to_shared = [&] (int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
      #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
          &sh_a_stage[a_sh_wr_trans[i]],
          &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
          a_sh_wr_pred[i]
        );
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
      #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
        cp_async4(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr], B_ptr[i]);
        B_ptr[i] += b_gl_rd_delta_o;
      }
      // Only fetch scales if this tile starts a new group
      if constexpr (group_blocks != -1) {
        if (pipe % (group_blocks / thread_k_blocks) == 0) {
          int4* sh_s3_stage = sh_s3 + s3_sh_stage * pipe;
          if (s3_sh_wr_pred)
            cp_async4(&sh_s3_stage[s3_sh_wr], &s3[s3_gl_rd]);
          s3_gl_rd += s3_gl_rd_delta;
        }
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&] () {
    // We only have `stages - 2` active fetches since we are double buffering and can only issue the next fetch when
    // it is guaranteed that the previous shared memory load is fully complete (as it may otherwise be overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe into the current register buffer.
  auto fetch_to_registers = [&] (int k, int pipe) {
    // It may seem inefficient that we reload the groups for every sub-tile; however, this does not seem to be a
    // significant bottleneck, while some theoretically better attempts have lead to bad instruction ordering by the
    // compiler and correspondingly a noticable drop in performance.
    if constexpr (group_blocks != -1) {
      int4* sh_s3_stage = sh_s3 + s3_sh_stage * ((group_blocks / thread_k_blocks) * (pipe / (group_blocks / thread_k_blocks)));
      reinterpret_cast<int4*>(&frag_s3[k % 2])[0] = sh_s3_stage[s3_sh_rd];
    }
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
    #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4(frag_a[k % 2][i], &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4* sh_b_stage = sh_b + b_sh_stage * pipe;
    frag_b_quant[k % 2] = *reinterpret_cast<I4*>(&sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd]);
  };

  // Execute the actual tensor core matmul of a sub-tile. 
  auto matmul = [&] (int k) {
    // We have the m dimension as the inner loop in order to encourage overlapping dequantization and matmul operations.
    #pragma unroll
    for (int j = 0; j < 4; j++) {
      int b_quant = frag_b_quant[k % 2][j];
      // int b_quant_shift = b_quant << 4;
      FragB frag_b0, frag_b1;
      // If there are no groups, we can just scale the final output once and can avoid doing so for each weight.
      if constexpr (group_blocks != -1) {
        int b_quant_shift = b_quant >> 8;
        frag_b0 = dequant_per_group(b_quant, frag_s3[k % 2][j], 0);
        frag_b1 = dequant_per_group(b_quant_shift, frag_s3[k % 2][j], 1);
      } else {
        int b_quant_shift = b_quant << 4;
        frag_b0 = dequant_per_channel(b_quant);
        frag_b1 = dequant_per_channel(b_quant_shift);
      }
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the number of warps while keeping the n
  // dimension of a tile reasonable, we have multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&] () {
    constexpr int red_off = threads / b_sh_stride / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride;
      constexpr int red_sh_stride = b_sh_stride * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride; 
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);

      // Parallel logarithmic shared memory reduction. We make sure to avoid any unnecessary read or write iterations,
      // e.g., for two warps we write only once by warp 1 and read only once by warp 0. 

      #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
        #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
            #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr = red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                int* c_rd = reinterpret_cast<int*>(&sh[red_sh_delta * j + red_sh_rd]);
                int* c_wr = reinterpret_cast<int*>(&sh[red_sh_wr]);
                #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] += c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] = reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
          #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            int* c_rd = reinterpret_cast<int*>(&sh[red_sh_delta * i + red_sh_rd]);
            #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] += c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we finally have to globally reduce over
  // the results. As the striped partioning minimizes the number of such reductions and our outputs are usually rather
  // small, we perform this reduction serially in L2 cache.
  // global_reduce works on INT32 elements, which are the results of INT8 GEMM.
  // This is why we need another INT32 maxtrix `C` to reduce instead of the
  // original half matrix `D`.
  auto global_reduce = [&] (bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to maximize L2 cache utilization in this step. 
    // To do this, we write out results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 4;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 8 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) + 8 * (threadIdx.x / 32) + (threadIdx.x % 4) * 2;
      c_gl_wr += (4 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads * 2;
      int c_sh_wr = 2 * threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
        // Interestingly, doing direct global accesses here really seems to mess up the compiler and lead to slowdowns,
        // hence we also use async-copies even though these fetches are not actually asynchronous.
        #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(
            &sh[c_sh_wr + c_sh_wr_delta * i],
            &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)],
            i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m
          );
          cp_async4_pred(
            &sh[c_sh_wr + c_sh_wr_delta * i + 1],
            &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2) + 1],
            i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m
          );
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

      #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 d_red1 = sh[c_sh_wr + i * c_sh_wr_delta];
            int4 d_red2 = sh[c_sh_wr + i * c_sh_wr_delta + 1];
            #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] += 
                reinterpret_cast<int*>(&d_red1)[j];
            }
            #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * (j + 4) + (i % 4)] += 
                reinterpret_cast<int*>(&d_red2)[j];
            }
          }
          if (!last) {
            int4 d1, d2;
            #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(&d1)[j] = 
                reinterpret_cast<int*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)];
            }
            #pragma unroll
            for (int j = 0; j < 4; j++) {
              reinterpret_cast<int*>(&d2)[j] = 
                reinterpret_cast<int*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * (j + 4) + (i % 4)];
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] = d1;
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2) + 1] = d2;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually reshuffle matrix fragments in this step,
  // the reduction above is performed in fragment layout. 
  auto write_result = [&] () {
    int d_gl_stride = prob_n / 8;
    constexpr int d_sh_stride = 2 * thread_n_blocks + 1;
    int d_gl_wr_delta = d_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int d_sh_rd_delta = d_sh_stride * (threads / (2 * thread_n_blocks));

    int d_gl_wr = d_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));
    d_gl_wr += (2 * thread_n_blocks) * slice_col;
    int d_sh_wr = (4 * d_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    d_sh_wr += 32 * (threadIdx.x / 32);
    int d_sh_rd = d_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));

    int d_gl_wr_end = d_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final global write patterns
    auto write = [&] (int idx, int c0, int c1, float a_s, FragS_CHANNEL& w_s) {
      float2 deq_res;
      deq_res.x = int32_to_float(c0) * w_s[0] * a_s;
      deq_res.y = int32_to_float(c1) * w_s[1] * a_s;
      ((half2*) sh)[idx] = float2_to_half2(deq_res);
    };

    if (threadIdx.x / 32 < thread_n_blocks / 4) {
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = d_sh_wr + 8 * j;
          write(wr + (4 * d_sh_stride) * 0 + 0, frag_c[i][j][0][0], frag_c[i][j][0][1], frag_s1[i][0], frag_s2[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * d_sh_stride) * 8 + 0, frag_c[i][j][0][2], frag_c[i][j][0][3], frag_s1[i][1], frag_s2[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * d_sh_stride) * 0 + 4, frag_c[i][j][1][0], frag_c[i][j][1][1], frag_s1[i][0], frag_s2[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * d_sh_stride) * 8 + 4, frag_c[i][j][1][2], frag_c[i][j][1][3], frag_s1[i][1], frag_s2[j / 2][2 * (j % 2) + 1]);
        }
        d_sh_wr += 16 * (4 * d_sh_stride);
      }
    }
    __syncthreads();

    #pragma unroll
    for (int i = 0; i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks)); i++) {
      if (d_gl_wr < d_gl_wr_end) {
        D[d_gl_wr] = sh[d_sh_rd];
        d_gl_wr += d_gl_wr_delta;
        d_sh_rd += d_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines. 
  auto start_pipes = [&] () {
    #pragma unroll
    for (int i = 0; i < stages - 1; i++)
      fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();

  // Main loop.
  while (slice_iters) {
    // We unroll over both the global fetch and the register load pipeline to ensure all shared memory accesses are
    // static. Note that both pipelines have even length meaning that the next iteration will always start at index 0.
    #pragma unroll
    for (int pipe = 0; pipe < stages;) {
      #pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe, slice_iters >= stages);
          pipe++;
          wait_for_stage();
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0)
        break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;

    // Process results and, if necessary, proceed to the next column slice. While this pattern may not be the most
    // readable, other ways of writing the loop seemed to noticeably worse performance after compliation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before write-out
      if (last) {
        if (s1_sh_wr_pred) {
          cp_async1(&sh_s1[s1_sh_wr], &s1[s1_gl_rd]);
        }
        if (s2_sh_wr_pred) {
          cp_async4(&sh_s2[s2_sh_wr], &s2[s2_gl_rd]);
        }
        cp_async_fence();
      }
      thread_block_reduce();
      if (last) {
        cp_async_wait<0>();
        __syncthreads();
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
          #pragma unroll
          for (int i = 0; i < thread_m_blocks; i++) {
            frag_s1[i][0] = *reinterpret_cast<float*>(&sh_s1[16 * i + 2 * s1_sh_rd]);
            frag_s1[i][1] = *reinterpret_cast<float*>(&sh_s1[16 * i + 2 * s1_sh_rd + 1]);
          }
          reinterpret_cast<int4*>(&frag_s2)[0] = sh_s2[s2_sh_rd + 0];
          reinterpret_cast<int4*>(&frag_s2)[1] = sh_s2[s2_sh_rd + 1];
          reinterpret_cast<int4*>(&frag_s2)[2] = sh_s2[s2_sh_rd + 8];
          reinterpret_cast<int4*>(&frag_s2)[3] = sh_s2[s2_sh_rd + 9];
        }
      }
      if (slice_count > 1) { // only globally reduce if there is more than one block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last) // only the last block in a slice actually writes the result
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
        #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
          #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++)
            B_ptr[i] -= b_gl_stride;
        }
        s3_gl_rd = s3_sh_stride * slice_col + threadIdx.x;
        s2_gl_rd = s2_sh_stride * slice_col + threadIdx.x;
        start_pipes();
      }
    }
  }
}




// 8 warps are a good choice since every SM has 4 schedulers and having more
// than 1 warp per schedule allows some more latency hiding. At the same time,
// we want relatively few warps to have many registers per warp and small tiles.
const int USER_THREADS = 256; // Note: This is only used with user-provided thread_k/n
const int STAGES = 4; // 4 pipeline stages fit into shared memory
// const int SHARED_MEM = 96 * 1024; // max shared memory on compute capability 8.6 (< 8.0)

static constexpr int min_thread_n = 64;
static constexpr int min_thread_k = 64;

static constexpr int tile_size = 16;
static constexpr int max_par = 16;

static constexpr int pack_factor_4bit =
    8;  // We have 8 4-bit vals inside a 32 bit

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256},  // Default
    {128, 64, 128},   // Reduce N 2X, same K
    {64, 256, 256},   // Reduce K 2X, increase N 2X
    {64, 128, 128},   // Reduce K 2X, same N
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},   // Default
    {128, 128, 256},  // Reduce N 2X, increase K 2X
    {64, 128, 128},   // Reduce N 2X, same K
    {128, 64, 128},   // Reduce N 4X, increase K 2X
};

bool is_valid_config(thread_config_t const& th_config, int prob_m, int prob_n,
                     int prob_k) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // thread_k can be only 128 or 64 (because it must be less than groupsize
  // which is 128)
  if (th_config.thread_k != 128 && th_config.thread_k != 64) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  return true;
}

thread_config_t determine_thread_config(int prob_m, int prob_n, int prob_k) {
  if (prob_m <= 16) {
    for (auto th_config : small_batch_thread_configs) {
      if (is_valid_config(th_config, prob_m, prob_n, prob_k)) {
        return th_config;
      }
    }

  } else {
    for (auto th_config : large_batch_thread_configs) {
      if (is_valid_config(th_config, prob_m, prob_n, prob_k)) {
        return th_config;
      }
    }
  }

  return thread_config_t{-1, -1, -1};
}

#define __CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,               \
                  GROUP_BLOCKS, NUM_THREADS)                                       \
  else if (thread_m_blocks == THREAD_M_BLOCKS &&                                   \
           thread_n_blocks == THREAD_N_BLOCKS &&                                   \
           thread_k_blocks == THREAD_K_BLOCKS &&                                   \
           group_blocks == GROUP_BLOCKS && num_threads == NUM_THREADS) {           \
    hipFuncSetAttribute(reinterpret_cast<const void*>(Marlin<NUM_THREADS), THREAD_M_BLOCKS, THREAD_N_BLOCKS,     \
                                THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>,            \
                         hipFuncAttributeMaxDynamicSharedMemorySize,              \
                         max_shared_mem);                                          \
    Marlin<NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,         \
           STAGES, GROUP_BLOCKS>                                                   \
        <<<blocks, NUM_THREADS, max_shared_mem, stream>>>(                         \
            A_ptr, B_ptr, C_ptr, D_ptr, s1_ptr, s2_ptr, s3_ptr,                    \
            prob_m, prob_n, prob_k, locks);                                        \
  }

#define CALL_IF(N_BLOCKS, K_BLOCKS, NUM_THREADS)    \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)

const int ERR_PROB_SHAPE = 1;
const int ERR_KERN_SHAPE = 2;

int qqq_cuda(
  const void* A,
  const void* B,
        void* C, // int32 reduce buffer
        void* D, // half
  const void* s1,
  const void* s2,
  const void* s3,
  int prob_m,
  int prob_n,
  int prob_k,
  void* workspace,
  int groupsize = -1,
  int dev = 0,
  hipStream_t stream = 0,
  int thread_k = -1,
  int thread_n = -1,
  int sms = -1,
  int max_par = 16
) {
  int tot_m = prob_m;
  int tot_m_blocks = ceildiv(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1)
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);

  // Set thread config
  thread_config_t th_config;
  if (thread_k != -1 && thread_n != -1) {
    // User-defined config
    th_config = thread_config_t{thread_k, thread_n, USER_THREADS};
  } else {
    // Auto config
    th_config = determine_thread_config(prob_m, prob_n, prob_k);
  }
  int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
  if (!is_valid_config(th_config, prob_m, prob_n, prob_k) || (group_blocks != -1 && prob_k % group_blocks != 0))
    return ERR_PROB_SHAPE;
  
  int num_threads = th_config.num_threads;
  thread_k = th_config.thread_k;
  thread_n = th_config.thread_n;
  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;
  int blocks = sms;

  if (groupsize == -1)
    assert(s3 == nullptr);
  if (prob_m == 0 || prob_n == 0 || prob_k == 0)
    return 0;
  
  const int4* A_ptr = (const int4*) A;
  const int4* B_ptr = (const int4*) B;
  int4* C_ptr = (int4*) C;
  int4* D_ptr = (int4*) D;
  const float* s1_ptr = (const float*) s1;
  const int4* s2_ptr = (const int4*) s2;
  const int4* s3_ptr = (const int4*) s3;

  int* locks = (int*) workspace;

  int ret = 0;
  for (int i = 0; i < tot_m_blocks; i += 4) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > 4) {
      // Note that parallel > 1 currently only works for inputs without any padding
      par = (16 * thread_m_blocks - pad) / 64;
      if (par > max_par)
        par = max_par;
      prob_m = 64 * par;
      i += 4 * (par - 1);
      thread_m_blocks = 4;
    }
    
    // For compilation speed, we only define the kernel configurations that have seemed useful (in terms of performance)
    // in our testing, however many more are, in principle, possible.
    if (false) {}
    CALL_IF(8, 8, 256)
    CALL_IF(16, 4, 256)
    CALL_IF(8, 4, 128)
    CALL_IF(4, 8, 128)
    else
      ret = ERR_KERN_SHAPE;

    A_ptr += 16 * thread_m_blocks * (prob_k / 16) * par;
    D_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
    s1_ptr += 16 * thread_m_blocks * par;
  }

  return ret;
}

void qqq_gemm(
  const torch::Tensor& A,
  const torch::Tensor& B,
        torch::Tensor& C,
        torch::Tensor& D,
  const torch::Tensor& s1,
  const torch::Tensor& s2,
  const torch::Tensor& s3,
        torch::Tensor& workspace,
  int thread_k = -1,
  int thread_n = -1,
  int sms = -1,
  int max_par = 8
) {
  int prob_m = A.size(0);
  int prob_n = C.size(1);
  int prob_k = A.size(1);
  int groupsize = (s3.numel() == 0) ? -1 : prob_k / s3.size(0);
  if (groupsize != -1 && groupsize * s3.size(0) != prob_k)
    AT_ERROR("k=", prob_k, " not compatible with ", s3.size(0), " groups.");
  if (workspace.numel() < prob_n / 128 * max_par)
    AT_ERROR("workspace must be of size at least ", prob_n / 128 * max_par, ".");
  if (s1.dtype() != torch::kFloat32)
     AT_ERROR("s1 dtype must be float32, but got ", s1.dtype(), ".");
  if (s2.dtype() != torch::kFloat32)
     AT_ERROR("s2 dtype must be float32, but got ", s2.dtype(), ".");
  if (s3.dtype() != torch::kFloat16)
     AT_ERROR("s3 dtype must be float16, but got ", s3.dtype(), ".");
  int dev = A.get_device();
  int err = qqq_cuda(
    A.data_ptr(),
    B.data_ptr(),
    C.data_ptr(),
    D.data_ptr(),
    s1.data_ptr(),
    s2.data_ptr(),
    s3.data_ptr(),
    prob_m, prob_n, prob_k,
    workspace.data_ptr(),
    groupsize,
    dev,
    at::cuda::getCurrentCUDAStream(dev),
    thread_k,
    thread_n,
    sms,
    max_par
  );

  if (err == ERR_PROB_SHAPE) {
    AT_ERROR(
      "Problem (m=", prob_m, ", n=", prob_n, ", k=", prob_k, ")",
      " not compatible with thread_k=", thread_k, ", thread_n=", thread_n, "."
    );
  } else if (err == ERR_KERN_SHAPE) {
    AT_ERROR(
      "No kernel implementation for thread_k=", thread_k, ", thread_n=", thread_n, ", groupsize=", groupsize, "."
    );
  }
}
