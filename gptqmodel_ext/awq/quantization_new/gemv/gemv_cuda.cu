#include "hip/hip_runtime.h"
/*
 * Modified from NVIDIA [TRT-LLM](https://github.com/NVIDIA/TensorRT-LLM/tree/d37b507f41a87457fe9f10f7459d08f5db235745/cpp/tensorrt_llm/kernels/weightOnlyBatchedGemv)
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
@article{lin2023awq,
  title={AWQ: Activation-aware Weight Quantization for LLM Compression and Acceleration},
  author={Lin, Ji and Tang, Jiaming and Tang, Haotian and Yang, Shang and Dang, Xingyu and Han, Song},
  journal={arXiv},
  year={2023}
}
*/

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdio.h>
#include <torch/extension.h>
#include "gemv_cuda.h"
#include "../dequantize.cuh"
#include "../dispatch_utils.cuh"
#define PACK_FACTOR 8
#define WARP_SIZE 32
#define MEM_ACCESS_SIZE 128

// Reduce sum within the warp using the tree reduction algorithm.
template <int Num, int WarpSize, typename T>
__device__ __forceinline__ static void warp_reduce(T* psum, float (*out_smem)[Num * 4])
{
  // kInterleave = 4
      float fpsum[Num];
      #pragma unroll
      for (int i = 0; i < Num; ++i)
      {
          fpsum[i] = static_cast<float>(psum[i]);
      }

      #pragma unroll
      for (int i = 0; i < Num; ++i)
      {
          // T0 + T1 + T8 + T9 + T16 + T17 + T24 + T25 (kInterleave = 4)
          fpsum[i] += __shfl_xor_sync(~0, fpsum[i], 16);
          fpsum[i] += __shfl_xor_sync(~0, fpsum[i], 8);
          fpsum[i] += __shfl_xor_sync(~0, fpsum[i], 1);
      }
      __syncthreads();
      int warp = threadIdx.x / WarpSize, lane = threadIdx.x % WarpSize;
      if (lane == 0 || lane == 2 || lane == 4 || lane == 6)
      {
          #pragma unroll
          for (int i = 0; i < Num; ++i)
          {
              out_smem[warp][i * 4 + lane / 2] = fpsum[i];
          }
      }
      __syncthreads();
};

__device__ __forceinline__ int make_divisible(int c, int divisor){
  return (c + divisor - 1) / divisor;
}

template <int NPerBlock, int Batch, int BlockSize, int GroupSize, typename T>
__global__ void gemv_kernel(
  const T* inputs, const uint32_t* weight, const T* scales, const T* zeros, T* outputs, 
  const int IC, const int OC)
{
    const int kStride = 64;
    const int kElemsPerThread = MEM_ACCESS_SIZE / 4;
    const int kThreadsNumPerTile = kStride / kElemsPerThread;
    // assert(MEM_ACCESS_SIZE == 128);

    using T2 = typename std::conditional<
        std::is_same<T, half>::value,
        half2,
        nv_bfloat162
    >::type;

    static constexpr int kShuffleSize = 32;
    static constexpr int kShuffleBasicTile = 2;
    static constexpr int kShuffleContinous = 4;
    static constexpr int kShuffleStrided = 4;

    constexpr int Num = NPerBlock * Batch;
    constexpr int kInterleave = 4;

    T local_inputs[kElemsPerThread];
    uint32_t local_qweights[MEM_ACCESS_SIZE / 32];
    T half_weight_buffer[kElemsPerThread]; 
    T dequantized_weight[kElemsPerThread * NPerBlock];
    T local_scale[NPerBlock];
    T local_scaled_zeros[NPerBlock];

    T psum[Num];
    for (int i = 0; i < Num; ++i)
        psum[i] = static_cast<T>(0.f);
    
    // extern __shared__ uint8_t shmem[];
    // float(*out_smem)[Num * kInterleave] = reinterpret_cast<float(*)[Num * kInterleave]>(shmem);
    __shared__ float out_smem[BlockSize / WARP_SIZE * 2][Num * kInterleave];

    const int blk_row_offset = blockIdx.x * NPerBlock * kInterleave;
    const int thd_row_offset = (threadIdx.x / kThreadsNumPerTile) % kInterleave;
    const int act_k_offset = threadIdx.x / (kThreadsNumPerTile * kInterleave) * kStride
                               + (threadIdx.x % kThreadsNumPerTile) * kElemsPerThread;
    const int group_offset = act_k_offset / GroupSize;
    // TODO: use make_divisible
    const uint32_t* blk_weight_ptr = weight + blk_row_offset * IC / PACK_FACTOR;
    const T* scale_ptr = scales + blk_row_offset + thd_row_offset + group_offset * OC;
    const T* zeros_ptr = zeros + blk_row_offset + thd_row_offset + group_offset * OC;
    const T* inputs_ptr = inputs + act_k_offset;

    const int act_forward_step = BlockSize * kElemsPerThread / kInterleave;
    const int scale_forward_step = act_forward_step / GroupSize * OC;

    // Main loop iteration, each block completes the outputs for several OCs
    for (int kk = threadIdx.x * kElemsPerThread; kk < IC * kInterleave; kk += BlockSize * kElemsPerThread)
    {
        // Load qweight, scales and scaled_zeros
        #pragma unroll
        for (int idx = 0; idx < NPerBlock; ++idx)
        {
            // use float4 to load weights, each thread load 32 int4 numbers (1 x float4, 128 bit)
            *((float4*)(local_qweights)) = 
                *((float4*)(blk_weight_ptr + (idx * kInterleave * IC + kk)/ PACK_FACTOR));
            local_scale[idx] = *(scale_ptr + idx * kInterleave);
            local_scaled_zeros[idx] = *(zeros_ptr + idx * kInterleave);
            
            // Map int4 qweight to fp format 
            #pragma unroll
            for (int i = 0; i < MEM_ACCESS_SIZE / 32; ++i)
            {
                // Converts 32 bits (8 x int4) to 8 fp16
                dequantize_s4_to_fp16x2<T>(*reinterpret_cast<half2 *>(local_qweights + i), reinterpret_cast<uint4 *>(half_weight_buffer + i * PACK_FACTOR));
            }

            // Dequantize (apply s/z) and shuffle elements to match the weight packing format
            #pragma unroll
            for (int i = 0; i < kShuffleContinous; ++i)
            {
                #pragma unroll
                for (int j = 0; j < kShuffleStrided; ++j)
                {
                    T2 w = 
                        *reinterpret_cast<T2*>(
                          half_weight_buffer + (i + j * kShuffleContinous)* kShuffleBasicTile
                        );
                    if constexpr (std::is_same<T, half>::value)
                    {
                      w = __hfma2(w, __half2half2(local_scale[idx]), __half2half2(local_scaled_zeros[idx]));
                    }
                    else
                    {
                      w = __hfma2(w, __bfloat162bfloat162(local_scale[idx]), __bfloat162bfloat162(local_scaled_zeros[idx]));
                    }
                    dequantized_weight[((i * kShuffleStrided + j) * kShuffleBasicTile + 0) 
                          * NPerBlock + idx]
                        = w.x;
                    dequantized_weight[((i * kShuffleStrided + j) * kShuffleBasicTile + 1)
                            * NPerBlock + idx]
                        = w.y;
                }
            }            
        }  
        #pragma unroll
        for (int batch_idx = 0; batch_idx < Batch; ++batch_idx)
        {
            const T* local_inputs_ptr = inputs_ptr + batch_idx * IC;
            #pragma unroll
            for (int idx = 0; idx < kElemsPerThread / 8; ++idx)
            {
                // load activation, 8 halves (128 bits) / step.
                *((float4*)(local_inputs + idx * 8)) = *((float4*)(local_inputs_ptr + idx * 8));
            }
            // Perform the MACs
            #pragma unroll
            for (int x = 0; x < NPerBlock / 2; ++x)
            {
                #pragma unroll
                for (int y = 0; y < kElemsPerThread; ++y)
                {
                    if constexpr (std::is_same<T, half>::value)
                    {                       
                      *reinterpret_cast<half2*>(psum + batch_idx * NPerBlock + x * 2)
                          = __hfma2(*reinterpret_cast<half2*>(dequantized_weight + y * NPerBlock + x * 2),
                              __half2half2(local_inputs[y]),
                              *reinterpret_cast<half2*>(psum + batch_idx * NPerBlock + x * 2));
                    }
                    else
                    {
                      *reinterpret_cast<nv_bfloat162*>(psum + batch_idx * NPerBlock + x * 2)
                          = __hfma2(*reinterpret_cast<nv_bfloat162*>(dequantized_weight + y * NPerBlock + x * 2),
                              __bfloat162bfloat162(local_inputs[y]),
                              *reinterpret_cast<nv_bfloat162*>(psum + batch_idx * NPerBlock + x * 2));   
                    }
                }
            }
        }
        inputs_ptr += act_forward_step;
        scale_ptr += scale_forward_step;
        zeros_ptr += scale_forward_step;
    }

    warp_reduce<Num, WARP_SIZE>(psum, out_smem);

    // Num * Interleave = batch * NPerBlock * Interleave -> 1 thread_block write back num
    for (int i = threadIdx.x; i < Num * kInterleave; i += BlockSize)
    {
        int batch_idx = i / (NPerBlock * kInterleave);
        int oc_idx = i % (NPerBlock * kInterleave);
        float acc = 0.f;
        for (int j = 0; j < BlockSize / WARP_SIZE; ++j)
        {
            acc += out_smem[j][i];
        }
        outputs[batch_idx * OC + blk_row_offset + oc_idx] = static_cast<T>(acc);
    }
}

/*
Computes GEMV (PyTorch interface).

Args:
  _in_feats: tensor of shape [B, IC];
  _kernel: int tensor of shape [OC, IC // 8];
  _zeros: int tensor of shape [OC, IC // G // 8];
  _scaling_factors: tensor of shape [OC, IC // G];
  blockDim_x: size of thread block, dimension x, where blockDim_x * workload_per_thread = IC;
  blockDim_y: size of thread block, dimension y, where blockDim_y * gridDim_y = OC;

Returns:
  out_feats: tensor of shape [B, OC];
*/
torch::Tensor gemv_forward_cuda_decode(
    torch::Tensor _in_feats,
    torch::Tensor _kernel,
    torch::Tensor _scaling_factors,
    torch::Tensor _zeros,
    int m,
    int n,
    int k,
    int group_size)
{

    std::vector<int64_t> output_shape = _in_feats.sizes().vec();
    output_shape.back() = n;

    auto data_type = _in_feats.scalar_type();
    TORCH_CHECK(_scaling_factors.scalar_type() == data_type);
    TORCH_CHECK(_zeros.scalar_type() == data_type);

    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
    at::Tensor _out_feats = torch::empty(output_shape, options);

    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(data_type, ctype, {
      auto in_feats = reinterpret_cast<ctype*>(_in_feats.data_ptr());
      auto kernel = reinterpret_cast<uint32_t*>(_kernel.data_ptr());
      auto zeros = reinterpret_cast<ctype*>(_zeros.data_ptr());
      auto scaling_factors = reinterpret_cast<ctype*>(_scaling_factors.data_ptr());
      auto out_feats = reinterpret_cast<ctype*>(_out_feats.data_ptr());
      
      static constexpr int N_PER_BLOCK = 2;
      static constexpr int K_INTERLEAVE = 4;
      static constexpr int BLOCK_SIZE = 256;

      dim3 num_blocks(n / N_PER_BLOCK / K_INTERLEAVE);
      dim3 num_threads(BLOCK_SIZE);

      // if (group_size == 64)
      // {
      //   gemv_kernel_g64<<<num_blocks, num_threads>>>(
      //     // pointers
      //     in_feats, kernel, zeros, scaling_factors, out_feats,
      //     // constants
      //     num_in_channels, num_out_channels
      //   );
      // }
      if (group_size == 128)
      {
        switch (m)
        {
        case 1:
          gemv_kernel<N_PER_BLOCK, 1, BLOCK_SIZE, 128><<<num_blocks, num_threads>>>(
            in_feats, kernel, scaling_factors, zeros, out_feats, k, n
          );
          break;
        case 2:
          gemv_kernel<N_PER_BLOCK, 2, BLOCK_SIZE, 128><<<num_blocks, num_threads>>>(
            in_feats, kernel, scaling_factors, zeros, out_feats, k, n
          );
          break;
        case 3:
          gemv_kernel<N_PER_BLOCK, 3, BLOCK_SIZE, 128><<<num_blocks, num_threads>>>(
            in_feats, kernel, scaling_factors, zeros, out_feats, k, n
          );
          break;
        case 4:
          gemv_kernel<N_PER_BLOCK, 4, BLOCK_SIZE, 128><<<num_blocks, num_threads>>>(
            in_feats, kernel, scaling_factors, zeros, out_feats, k, n
          );
          break;
        case 5:
          gemv_kernel<N_PER_BLOCK, 5, BLOCK_SIZE, 128><<<num_blocks, num_threads>>>(
            in_feats, kernel, scaling_factors, zeros, out_feats, k, n
          );
          break;
        case 6:
          gemv_kernel<N_PER_BLOCK, 6, BLOCK_SIZE, 128><<<num_blocks, num_threads>>>(
            in_feats, kernel, scaling_factors, zeros, out_feats, k, n
          );
          break;
        case 7:
          gemv_kernel<N_PER_BLOCK, 7, BLOCK_SIZE, 128><<<num_blocks, num_threads>>>(
            in_feats, kernel, scaling_factors, zeros, out_feats, k, n
          );
          break;
        default:
          throw std::runtime_error("Unsupported batch size for gemv kernel.\n");
        }
      }
      else
      {
        throw std::runtime_error("Unsupported group size for gemv kernel.\n");
      }
    });
    return _out_feats;
}
