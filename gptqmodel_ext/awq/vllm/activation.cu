#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#define VLLM_LDG(arg) *(arg)

#define VLLM_DISPATCH_CASE_FLOATING_TYPES(...)              \
  AT_DISPATCH_CASE(at::ScalarType::Float, __VA_ARGS__)      \
  AT_DISPATCH_CASE(at::ScalarType::Half, __VA_ARGS__)       \
  AT_DISPATCH_CASE(at::ScalarType::BFloat16, __VA_ARGS__)

#define VLLM_DISPATCH_FLOATING_TYPES(TYPE, NAME, ...)             \
  AT_DISPATCH_SWITCH(                                             \
    TYPE, NAME, VLLM_DISPATCH_CASE_FLOATING_TYPES(__VA_ARGS__))

template<typename T>
__device__ __forceinline__ T silu(const T& x) {
  // x * sigmoid(x)
  return (T) (((float) x) / (1.0f + expf((float) -x)));
}

template<typename scalar_t>
__global__ void silu_and_mul_kernel(
  scalar_t* __restrict__ out,               // [..., d]
  const scalar_t* __restrict__ input,       // [..., 2, d]
  const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = VLLM_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = silu(x) * y;
  }
}


void silu_and_mul(
  torch::Tensor& out,      // [..., d]
  torch::Tensor& input)    // [..., 2 * d]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int d = input.size(-1) / 2;

  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    input.scalar_type(),
    "silu_and_mul_kernel",
    [&] {
      silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        d);
    });
}